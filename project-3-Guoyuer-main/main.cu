#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#define MAX(X, Y) (((X) > (Y)) ? (X) : (Y))
#define MIN(a, b) (((a) < (b)) ? (a) : (b))

#define M_PI 3.14159265358979323846 /* pi */
#define MAX_BLOCKS_PER_DIM 2147483647
#define MAX_RAND_NUM 200


typedef struct vector
{
  double x;
  double y;
  double z;
} vector;

__device__ uint64_t fast_forward_LCG(uint64_t seed, uint64_t n);
__device__ double LCG_random_double(uint64_t *seed);

__device__ vector scale(vector v, double s)
{
  vector nv;
  nv.x = v.x * s;
  nv.y = v.y * s;
  nv.z = v.z * s;
  return nv;
}

__device__ vector subtract(vector a, vector b)
{
  vector nv;
  nv.x = a.x - b.x;
  nv.y = a.y - b.y;
  nv.z = a.z - b.z;
  return nv;
}

__device__ double product(vector a, vector b)
{
  double sum = 0;
  sum += a.x * b.x;
  sum += a.y * b.y;
  sum += a.z * b.z;
  return sum;
}

__device__ double length(vector v)
{
  return sqrt(product(v, v));
}
__device__ vector normalize(vector v)
{
  double len = length(v);
  return scale(v, 1.0 / len);
}

void print(vector v)
{
  printf("(%.2f, %.2f, %.2f)\n", v.x, v.y, v.z);
}

__global__ void fill_vec(vector *v, double x, double y, double z)
{
  v->x = x;
  v->y = y;
  v->z = z;
}
__device__ int check(vector W, double Wmax, double R, double term);

__global__ void compute(double *G, int n_ray, int n_grid_1d, vector L, vector C,
                        double R, double Wy, double Wmax)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  // two seeds, for two distribution
  
  uint64_t init_seed1 = (uint64_t)tid * 3153 + 451028;
  uint64_t seed1 = fast_forward_LCG(init_seed1, tid * MAX_RAND_NUM);

  uint64_t init_seed2 = (uint64_t)tid * 784031 + 145648;
  uint64_t seed2 = fast_forward_LCG(init_seed2, tid * MAX_RAND_NUM);
  vector I, N, S;
  // grid-stride loop
  for(int n = 0; n < n_ray; n += blockDim.x * gridDim.x)
  {
    vector W, V;
    double term;
    do
    {
      double phi = LCG_random_double(&seed1) * 2 * M_PI;
      double cos_th = -1 + LCG_random_double(&seed2) * 2;
      double sin_th = sqrt(1 - cos_th * cos_th);
      double Vx = sin_th * cos(phi);
      double Vy = sin_th * sin(phi);
      double Vz = cos_th;
      V.x = Vx;
      V.y = Vy;
      V.z = Vz;
      W = scale(V, Wy / Vy);
      term = pow(product(V, C), 2) + pow(R, 2) - product(C, C);
    } while (!check(W, Wmax, R, term));
    double t = product(V, C) - sqrt(term);
    I = scale(V, t);
    vector I_C = subtract(I, C);
    N = normalize(I_C);
    vector L_I = subtract(L, I);
    S = normalize(L_I);
    double b = MAX(0.0, product(S, N));
    int i = W.x / (Wmax / (n_grid_1d / 2)) + n_grid_1d / 2;
    int j = W.z / (Wmax / (n_grid_1d / 2)) + n_grid_1d / 2;
    int idx = (n_grid_1d - 1 - i) * n_grid_1d + j; // flatten the 2d matrix;
    atomicAdd(&G[idx], b);
  }
}

__device__ int check(vector W, double Wmax, double R, double term)
{
  return (abs(W.x) < Wmax && abs(W.z) < Wmax && term > 0);
}
void save_matrix(double *A, int N, char *fname)
{
  FILE *fp = fopen(fname, "w");

  for (long i = 0; i < N; i++)
  {
    for (long j = 0; j < N; j++)
    {
      long idx = i * N + j;
      fprintf(fp, "%.7le ", A[idx]);
    }
    fprintf(fp, "\n");
  }
  fclose(fp);
}

__device__ uint64_t fast_forward_LCG(uint64_t seed, uint64_t n)
{
  const uint64_t m = 9223372036854775808ULL;
  uint64_t a = 2806196910506780709ULL;
  uint64_t c = 1ULL;
  n = n % m;
  uint64_t a_new = 1;
  uint64_t c_new = 0;

  while (n > 0)
  {
    if (n & 1)
    {
      a_new *= a;
      c_new = c_new * a + c;
    }
    c *= (a + 1);
    a *= a;
    n >>= 1;
  }
  return (a_new * seed + c_new) % m;
}

__device__ double LCG_random_double(uint64_t *seed)
{
  const uint64_t m = 9223372036854775808ULL;
  const uint64_t a = 2806196910506780709ULL;

  const uint64_t c = 1ULL;
  *seed = (a * (*seed) + c) % m;
  return (double)(*seed) / (double)m;
}

int main(int argc, char *argv[])
{

  int n_ray = atoi(argv[1]);
  int n_grid_1d = atoi(argv[2]);
  int NTHREADS_PER_BLOCK = atoi(argv[3]);
  if(NTHREADS_PER_BLOCK > 1024){
    printf("The maximum number of thread per block is 1024!\n");
    return 0;
  }
  vector L = {4, 4, -1};
  vector C = {0, 12, 0};
  double R = 6, Wy = 10, Wmax = 10;
  size_t len = n_grid_1d * n_grid_1d * sizeof(double);
  double *G_h = (double *)malloc(len);
  double *G;
  hipMalloc((void **)&G, len);
  int nblocks = MIN(n_ray / NTHREADS_PER_BLOCK + 1, MAX_BLOCKS_PER_DIM);
  printf("nblocks: %d;  NTHREADS_PER_BLOCK: %d; avg_nrays_per_thread %.2f\n", nblocks, NTHREADS_PER_BLOCK, (double) n_ray/(nblocks*NTHREADS_PER_BLOCK));
  // if (nblocks == MAX_BLOCKS_PER_DIM)
  // {
  //   printf("Total number of thread < number of rays!\n");
  //   printf("Still works!\n");
  // }
  hipDeviceSynchronize();

  hipEvent_t start_device, stop_device;
  hipEventCreate(&start_device);
  hipEventCreate(&stop_device);
  float time_device;
  hipEventRecord(start_device, 0);

  compute<<<nblocks, NTHREADS_PER_BLOCK>>>(G, n_ray, n_grid_1d, L, C, R, Wy, Wmax);

  hipEventRecord(stop_device, 0);
  hipEventSynchronize(stop_device);
  hipEventElapsedTime(&time_device, start_device, stop_device);
  printf("time elapsed on GPU: %f(s)\n", time_device / 1000.);
  hipEventDestroy(start_device);
  hipEventDestroy(stop_device);

  hipDeviceSynchronize();

  hipMemcpy(G_h, G, len, hipMemcpyDeviceToHost);

  save_matrix(G_h, n_grid_1d, "window.out");
  return 0;
}